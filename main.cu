#include "hip/hip_runtime.h"
#include <memory>
#include <string>
#include <stdexcept>
#include <vector>
#include <chrono>
#include <iostream>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

constexpr size_t n_block    = 64;
constexpr size_t n_thread   = 128;
constexpr size_t n_iter     = 5;
constexpr size_t n_rep      = 10;
constexpr size_t n_element_lo  =   512;
constexpr size_t n_element_hi  = 65536;
constexpr size_t n_epoch_lo = 128;
constexpr size_t n_epoch_hi = 128;
constexpr size_t n_chunk_lo = 32;
constexpr size_t n_chunk_hi = 512;

using timer = std::chrono::high_resolution_clock;
using nsecs = std::chrono::microseconds;

namespace kernels {
  __global__ void empty(size_t n) {}

  __global__ void axpy(double *y, double* x, double alpha, size_t n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i<n) { y[i] += alpha*x[i]; }
  }

  __device__ double f(double x)  { return exp(cos(x)) - 2; }
  __device__ double fp(double x) { return -sin(x) * exp(cos(x)); }

  __global__ void newton(size_t n_iter, double *x, size_t n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < n) {
      auto x0 = x[i];
      for(int iter = 0; iter < n_iter; ++iter) {
	x0 -= f(x0)/fp(x0);
      }
      x[i] = x0;
    }
  }
}

using result = std::vector<double>;

#define cuda_api(error, f, ...)						\
  do {									\
    auto rc = f(__VA_ARGS__);						\
    if (rc != hipSuccess) {						\
      throw std::runtime_error(std::string(error) + ": " +  hipGetErrorString(rc)); \
    } else {								\
      /*std::cerr << error << '\n';*/					\
    }									\
  } while (0)

void print_result(result& res, 
		  size_t epoch, size_t chunk, size_t element, 
		  const std::string& kind, const std::string& kernel) {
  std::cout << kind    << "," 
	    << kernel  << ","
	    << epoch   << ","
	    << chunk   << ","
	    << element << ",";
  for (const auto& r: res)  std::cout << r << ",";    
  std::cout << std::endl;
  std::cout.flush();
}						

auto bench_null() {
  result res;
  for (auto rep = 0; rep < n_rep; ++rep) {
    auto t0 = timer::now();
    cuda_api("pre-sync", hipDeviceSynchronize);
    cuda_api("post-sync", hipDeviceSynchronize);
    auto t1 = timer::now();
    auto dt = (t1 - t0).count();
    res.push_back(dt);
  }
  return res;
}

template<typename K, typename... As>
auto bench_kernels(const std::string& tag, size_t n_epoch, size_t n_chunk, size_t n_element, K kernel, As... as) {
  result res;
  for (auto rep = 0; rep < n_rep; ++rep) {
    auto t0 = timer::now();
    cuda_api("pre-sync", hipDeviceSynchronize);
    for (auto epoch = 0ul; epoch < n_epoch; ++epoch) {
      for (auto serial = 0ul; serial < n_chunk; ++serial) {
	kernel<<<n_thread, n_block>>>(as...);
      }
      for (auto parallel = 0ul; parallel < n_chunk; ++parallel) {
	kernel<<<n_thread, n_block>>>(as...);
      }
    }
    cuda_api("post-sync", hipDeviceSynchronize);
    auto t1 = timer::now();
    auto dt = (t1 - t0).count();
    res.push_back(dt);
  }
  print_result(res, n_epoch, n_chunk, n_element, tag, "kernels");
}

hipGraphNode_t add_empty_node(hipGraph_t& graph) {
  hipGraphNode_t node = {0};
  cuda_api("add empty", hipGraphAddEmptyNode, &node, graph, nullptr, 0);
  return node;
}

template<class K>
hipGraphNode_t add_kernel_node(hipGraph_t& graph, K kernel, const std::vector<void*> args) {
  hipGraphNode_t node = {0};
  hipKernelNodeParams params = {0};
  params.func           = (void*) kernel;
  params.gridDim        = n_block;
  params.blockDim       = n_thread;
  params.sharedMemBytes = 0;
  params.kernelParams   = (void**) args.data();
  params.extra          = nullptr;

  cuda_api("kernel add", hipGraphAddKernelNode, &node, graph, nullptr, 0, &params);

  return node;
}

template<int N, int ix, typename... Ts>
constexpr void set_args(std::unique_ptr<std::tuple<Ts...>>& values, std::vector<void*>& pointers) {
  if constexpr(ix >= N) {
      return;
    } else {
    pointers[ix] = &std::get<ix>(*values);
    set_args<N, ix + 1, Ts...>(values, pointers);
  }
}

void add_dependencies(hipGraph_t& graph, const hipGraphNode_t& from, const hipGraphNode_t& to) {
  cuda_api("add edge", hipGraphAddDependencies, graph, &from, &to, 1); 
}

void add_dependencies(hipGraph_t& graph, const hipGraphNode_t& from_, const std::vector<hipGraphNode_t>& to) {
  auto n = to.size();
  std::vector<hipGraphNode_t> from(n, from_);
  cuda_api("add edge", hipGraphAddDependencies, graph, from.data(), to.data(), n); 
}

void add_dependencies(hipGraph_t& graph, const std::vector<hipGraphNode_t>& from, const hipGraphNode_t& to_) {
  auto n = from.size();
  std::vector<hipGraphNode_t> to(n, to_);
  cuda_api("add edge", hipGraphAddDependencies, graph, from.data(), to.data(), n); 
}

template<typename K, typename... As>
auto bench_graph(const std::string& tag, size_t n_epoch, size_t n_chunk, size_t n_element, K kernel, As... as) {
  hipStream_t    stream   = {0};
  hipGraph_t     graph    = {0};
  hipGraphExec_t instance = {0};

  std::vector<void*> args(sizeof...(As), nullptr);
  auto tmp = std::make_unique<std::tuple<As...>>(as...);
  set_args<sizeof...(As), 0, As...>(tmp, args);

  cuda_api("stream init", hipStreamCreate, &stream);
  cuda_api("graph init", hipGraphCreate, &graph, 0);

  auto root = add_empty_node(graph);
  auto last = root;

  for (auto epoch = 0ul; epoch < n_epoch; ++epoch) {
    for (auto serial = 0ul; serial < n_chunk; ++serial) {
      auto node = add_kernel_node(graph, kernel, args);
      add_dependencies(graph, last, node);
      last = node;
    }
    std::vector<hipGraphNode_t> nodes;
    for (auto parallel = 0ul; parallel < n_chunk; ++parallel) {
      auto node = add_kernel_node(graph, kernel, args);
      nodes.push_back(std::move(node));
    }
    last = add_empty_node(graph);
    add_dependencies(graph, last, nodes);
    last = add_empty_node(graph);
    add_dependencies(graph, nodes, last);
  }
  cuda_api("instantiate graph", hipGraphInstantiate, &instance, graph, nullptr, nullptr, 0);
  result res;
  for (auto rep = 0; rep < n_rep; ++rep) {
    auto t0 = timer::now();
    cuda_api("pre-sync", hipDeviceSynchronize);
    cuda_api("graph exec", hipGraphLaunch, instance, stream);
    cuda_api("post-sync", hipDeviceSynchronize);
    auto t1 = timer::now();
    auto dt = (t1 - t0).count();
    res.push_back(dt);
  }
  print_result(res, n_epoch, n_chunk, n_element, tag, "graphs");
}

int main() {
  double* x;
  double* y;
  double alpha;

  cuda_api("malloc x", hipMalloc, &x, n_element_hi);
  cuda_api("malloc y", hipMalloc, &y, n_element_hi);

  for (auto n_epoch = n_epoch_lo; n_epoch <= n_epoch_hi; n_epoch *= 2) {
    for (auto n_chunk = n_chunk_lo; n_chunk <= n_chunk_hi; n_chunk *= 2) {
      for (auto n_element = n_element_lo; n_element <= n_element_hi; n_element *= 2) {
	bench_kernels("empty",  n_epoch, n_chunk, n_element, kernels::empty, n_element);
	bench_kernels("axpy",   n_epoch, n_chunk, n_element, kernels::axpy, y, x, alpha, n_element);
	bench_kernels("newton", n_epoch, n_chunk, n_element, kernels::newton, n_iter, x, n_element);
	bench_graph("empty",    n_epoch, n_chunk, n_element, kernels::empty, n_element);
	bench_graph("axpy",     n_epoch, n_chunk, n_element, kernels::axpy, y, x, alpha, n_element);
	bench_graph("newton",   n_epoch, n_chunk, n_element, kernels::newton, n_iter, x, n_element);
      }
    }
  }
}
