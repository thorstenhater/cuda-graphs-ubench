#include "hip/hip_runtime.h"
#include <memory>
#include <string>
#include <stdexcept>
#include <vector>
#include <chrono>
#include <iostream>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

constexpr size_t n_block    = 64;
constexpr size_t n_thread   = 128;
constexpr size_t n_iter     = 5;
constexpr size_t n_rep      = 10;
constexpr size_t n_element  = 65336;


struct benchmark_parameters {
 size_t n_epoch;
 size_t n_serial;
 size_t n_parallel;
};

struct benchmark_range {
  benchmark_parameters from, to, current;
  benchmark_parameters& operator*() {
    return current;
  }

  benchmark_range& operator++() {
    current.n_serial *= 2;
    current.n_parallel *= 2;
    return *this;
  }
};

using timer = std::chrono::high_resolution_clock;
using nsecs = std::chrono::microseconds;

namespace kernels {
  __global__ void empty(size_t n) {}

  __global__ void axpy(double *y, double* x, double alpha, size_t n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i<n) { y[i] += alpha*x[i]; }
  }

  __device__ double f(double x)  { return exp(cos(x)) - 2; }
  __device__ double fp(double x) { return -sin(x) * exp(cos(x)); }

  __global__ void newton(size_t n_iter, double *x, size_t n) {
    auto i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < n) {
      auto x0 = x[i];
      for(int iter = 0; iter < n_iter; ++iter) {
	x0 -= f(x0)/fp(x0);
      }
      x[i] = x0;
    }
  }
}

using result = std::vector<double>;

#define cuda_api(error, f, ...)						\
  do {									\
    auto rc = f(__VA_ARGS__);						\
    if (rc != hipSuccess) {						\
      throw std::runtime_error(std::string(error) + ": " +  hipGetErrorString(rc)); \
    } else {								\
      /*std::cerr << error << '\n';*/					\
    }									\
  } while (0)

void print_result(result& res, size_t n, const std::vector<std::string> tags) {
  for (const auto& t: tags) std::cout << t << " ";    
  std::cout << "," << n << ",";    
  for (const auto& r: res)  std::cout << r << ",";    
  std::cout << '\n';				
}						

auto bench_null() {
  result res;
  for (auto rep = 0; rep < n_rep; ++rep) {
    auto t0 = timer::now();
    cuda_api("pre-sync", hipDeviceSynchronize);
    cuda_api("post-sync", hipDeviceSynchronize);
    auto t1 = timer::now();
    auto dt = (t1 - t0).count();
    res.push_back(dt);
  }
  return res;
}

template<typename K, typename... As>
auto bench_kernels(const benchmark_parameters& p, K kernel, As... as) {
  result res;
  for (auto rep = 0; rep < n_rep; ++rep) {
    auto t0 = timer::now();
    cuda_api("pre-sync", hipDeviceSynchronize);
    for (auto epoch = 0ul; epoch < p.n_epoch; ++epoch) {
      for (auto serial = 0ul; serial < p.n_serial; ++serial) {
	kernel<<<n_thread, n_block>>>(as...);
      }
      for (auto parallel = 0ul; parallel < p.n_parallel; ++parallel) {
	kernel<<<n_thread, n_block>>>(as...);
      }
    }
    cuda_api("post-sync", hipDeviceSynchronize);
    auto t1 = timer::now();
    auto dt = (t1 - t0).count();
    res.push_back(dt);
  }
  return res;
}

hipGraphNode_t add_empty_node(hipGraph_t& graph) {
  hipGraphNode_t node = {0};
  cuda_api("add empty", hipGraphAddEmptyNode, &node, graph, nullptr, 0);
  return node;
}

template<class K>
hipGraphNode_t add_kernel_node(hipGraph_t& graph, K kernel, const std::vector<void*> args) {
  hipGraphNode_t node = {0};
  hipKernelNodeParams params = {0};
  params.func           = (void*) kernel;
  params.gridDim        = n_block;
  params.blockDim       = n_thread;
  params.sharedMemBytes = 0;
  params.kernelParams   = (void**) args.data();
  params.extra          = nullptr;

  cuda_api("kernel add", hipGraphAddKernelNode, &node, graph, nullptr, 0, &params);

  return node;
}

template<int N, int ix, typename... Ts>
constexpr void set_args(std::unique_ptr<std::tuple<Ts...>>& values, std::vector<void*>& pointers) {
  if constexpr(ix >= N) {
      return;
    } else {
    pointers[ix] = &std::get<ix>(*values);
    set_args<N, ix + 1, Ts...>(values, pointers);
  }
}

void add_dependencies(hipGraph_t& graph, const hipGraphNode_t& from, const hipGraphNode_t& to) {
  cuda_api("add edge", hipGraphAddDependencies, graph, &from, &to, 1); 
}

void add_dependencies(hipGraph_t& graph, const hipGraphNode_t& from_, const std::vector<hipGraphNode_t>& to) {
  auto n = to.size();
  std::vector<hipGraphNode_t> from(n, from_);
  cuda_api("add edge", hipGraphAddDependencies, graph, from.data(), to.data(), n); 
}

void add_dependencies(hipGraph_t& graph, const std::vector<hipGraphNode_t>& from, const hipGraphNode_t& to_) {
  auto n = from.size();
  std::vector<hipGraphNode_t> to(n, to_);
  cuda_api("add edge", hipGraphAddDependencies, graph, from.data(), to.data(), n); 
}

template<typename K, typename... As>
auto bench_graph(const benchmark_parameters& p, K kernel, As... as) {
  hipStream_t    stream   = {0};
  hipGraph_t     graph    = {0};
  hipGraphExec_t instance = {0};

  std::vector<void*> args(sizeof...(As), nullptr);
  auto tmp = std::make_unique<std::tuple<As...>>(as...);
  set_args<sizeof...(As), 0, As...>(tmp, args);

  cuda_api("stream init", hipStreamCreate, &stream);
  cuda_api("graph init", hipGraphCreate, &graph, 0);

  auto root = add_empty_node(graph);
  auto last = root;

  for (auto epoch = 0ul; epoch < p.n_epoch; ++epoch) {
    for (auto serial = 0ul; serial < p.n_serial; ++serial) {
      auto node = add_kernel_node(graph, kernel, args);
      add_dependencies(graph, last, node);
      last = node;
    }
    std::vector<hipGraphNode_t> nodes;
    for (auto parallel = 0ul; parallel < p.n_parallel; ++parallel) {
      auto node = add_kernel_node(graph, kernel, args);
      nodes.push_back(std::move(node));
    }
    last = add_empty_node(graph);
    add_dependencies(graph, last, nodes);
    last = add_empty_node(graph);
    add_dependencies(graph, nodes, last);
  }
  cuda_api("instantiate graph", hipGraphInstantiate, &instance, graph, nullptr, nullptr, 0);
  result res;
  for (auto rep = 0; rep < n_rep; ++rep) {
    auto t0 = timer::now();
    cuda_api("pre-sync", hipDeviceSynchronize);
    cuda_api("graph exec", hipGraphLaunch, instance, stream);
    cuda_api("post-sync", hipDeviceSynchronize);
    auto t1 = timer::now();
    auto dt = (t1 - t0).count();
    res.push_back(dt);
  }
  return res;
}

int main() {
  double* x;
  double* y;
  double alpha;

  cuda_api("alloc x", hipMalloc, &x, n_element);
  cuda_api("alloc y", hipMalloc, &y, n_element);

  {
    auto res = bench_null();
    print_result(res, -1, {"null"});
  }

  for (size_t c = 
  for (size_t n = 128; n <= n_element; n *= 2) {
    {
      auto res = bench_kernels(kernels::empty, n);
      print_result(res, n, {"kernel", "empty"});
    }
    {
      auto res = bench_kernels(kernels::axpy, y, x, alpha, n);
      print_result(res, n, {"kernel", "axpy"});
    }
    {
      auto res = bench_kernels(kernels::newton, n_iter, x, n);
      print_result(res, n, {"kernel", "newton"});
    }
    {
      auto res = bench_graph(kernels::empty, n);
      print_result(res, n, {"graph", "empty"});
    }
    {
      auto res = bench_graph(kernels::axpy, y, x, alpha, n);
      print_result(res, n, {"graph", "axpy"});
    }
    {
      auto res = bench_graph(kernels::newton, n_iter, x, n);
      print_result(res, n, {"graph", "newton"});
    }
  }
}
